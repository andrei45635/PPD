#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <time.h>
#include <chrono>

__global__
void convolutionKernel(int* matrix, const double* kernel, int n, int m, int kernel_size, int startRow, int endRow) {
    //cooperative_groups::grid_group g = cooperative_groups::this_grid();

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= startRow && row <= endRow && col < m) {
        double temp = 0.0;
        int kernel_radius = kernel_size / 2;
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            for (int j = -kernel_radius; j <= kernel_radius; j++) {
                int r = __max(0, __min(row + i, n - 1));
                int c = __max(0, __min(col + j, m - 1));
                temp += matrix[r * m + c] * kernel[(i + kernel_radius) * kernel_size + (j + kernel_radius)];
            }
        }
        __syncthreads();
        matrix[row * m + col] = static_cast<int>(temp);
    }

    //g.sync();
}

void readMatrixFromFile(const std::string& filename, std::vector<int>& matrix, int& rows, int& cols) {
    std::ifstream file(filename);
    if (!file) {
        std::cerr << "Error opening file: " << filename << std::endl;
        exit(1);
    }
    
    file >> rows >> cols;
    matrix.resize(rows * cols);
    
    for (int i = 0; i < rows * cols; i++) {
        file >> matrix[i];
    }
}

void writeMatrixToFile(const std::string& filename, const std::vector<int>& matrix, int rows, int cols) {
    std::ofstream file(filename);
    if (!file) {
        std::cerr << "Error opening file for writing: " << filename << std::endl;
        exit(1);
    }

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            file << matrix[i * cols + j] << " ";
        }
        file << "\n";
    }

    file.close();
}

int main()
{
    std::string small_matrix = "C:\\Users\\GIGABYTE\\IdeaProjects\\PPD\\ppd_lab2_tema\\src\\main\\resources\\date.txt";
    std::string huge_matrix = "C:\\Users\\GIGABYTE\\IdeaProjects\\PPD\\ppd_lab2_tema\\src\\main\\resources\\lab3_input.txt";
    std::string huge_matrix_2 = "C:\\Users\\GIGABYTE\\IdeaProjects\\PPD\\ppd_lab2_tema\\src\\main\\resources\\lab3_marimi.txt";
    //double kernel[3][3] = { 5, 1, 15, 21, 12, 43, 52, 69, 88 };
    double kernel[3][3] = { 0, 1, 0, 0, 0, 0, 0, 0, 0 };
    int n = 1000, m = 1000;
    int rows, cols;
    std::vector<int> matrix;
    
    readMatrixFromFile(huge_matrix_2, matrix, rows, cols);

    auto t_start = std::chrono::high_resolution_clock::now();
    
    int* dev_matrix;
    size_t matrix_size = rows * cols * sizeof(int);
    hipMalloc(&dev_matrix, matrix_size);
    hipMemcpy(dev_matrix, matrix.data(), matrix_size, hipMemcpyHostToDevice);

    double* dev_kernel;
    size_t kernel_size = 3 * 3 * sizeof(double);
    hipMalloc(&dev_kernel, kernel_size);
    hipMemcpy(dev_kernel, kernel, kernel_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);

    int rowsPerSegment = 50;
    int totalSegments = (n / 2) / rowsPerSegment;
    
    for (int seg = 0; seg < totalSegments; seg++) {
        int startRow = seg * rowsPerSegment;
        int endRow = seg * rowsPerSegment;

        dim3 blocksPerGrid((m + threadsPerBlock.x - 1) / threadsPerBlock.x, (rowsPerSegment + threadsPerBlock.y - 1) / threadsPerBlock.y);

        convolutionKernel<<<blocksPerGrid, threadsPerBlock>>>(dev_matrix, dev_kernel, n, m, 3, startRow, endRow);
        hipDeviceSynchronize();
    }

    hipMemcpy(matrix.data(), dev_matrix, matrix_size, hipMemcpyDeviceToHost);

    // Check for any errors in kernel launch
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "convolutionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    // Wait for GPU to finish before accessing on host
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching convolutionKernel!\n", cudaStatus);
        return 1;
    }

    // Copy result back to host
    hipMemcpy(matrix.data(), dev_matrix, matrix_size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(dev_matrix);
    hipFree(dev_kernel);
    
    auto t_end = std::chrono::high_resolution_clock::now();
    double difference = std::chrono::duration<double, std::milli>(t_end - t_start).count();

    std::cout << difference << std::endl;

    std::string output = "C:\\Users\\GIGABYTE\\IdeaProjects\\PPD\\ppd_lab2_tema\\src\\main\\resources\\output.txt";
    writeMatrixToFile(output, matrix, rows, cols);

    return 0;
}
