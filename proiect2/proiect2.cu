#include <hip/hip_runtime.h>


#include<iostream>
#include<fstream>
#include<cstdlib>
#include<chrono>
#include<condition_variable>
#include<vector>

#define KERNEL_SIZE 3
#define MAX_THREADS_PER_BLOCK 1024

typedef struct{
    int * L_frontier;
    int * R_frontier;
}side_frontiers;

__device__ __host__
int clip(int x, int a, int b){
	// clips x between a and b
	return x < a ? a : (x > b ? b : x);
}

void cudaCheckError(hipError_t cudaResult){
    if (cudaResult != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaResult) << std::endl;
        exit(1);
    }
}

__global__
void convolution(
int N, int start, int end,
int ** matrix, int ** kernel,
int * N_frontier, int * S_frontier,
int * current_line, // stores the index of the line in the matrix that is being processed currently
side_frontiers side_frontiers,
int grid_index
){
    
    int i = start + blockIdx.x;
    int j = MAX_THREADS_PER_BLOCK * grid_index + threadIdx.x;

    // if(threadIdx.x == 0 && blockIdx.x == 0){
    //     printf("%d %d %d %d %d %d\n", start, end, i, j, *current_line, blockDim.x);
    // }

    while(i != atomicAdd(current_line, 0)); // spin lock until current_line is equal to the block's line

    int NW_value;
    int N_value;
    int NE_value;
    int W_value;
    int center_value;
    int E_value;
    int SW_value;
    int S_value;
    int SE_value;

    N_value      = atomicAdd(&N_frontier[j], 0);
    center_value = atomicAdd(&matrix[i][j], 0);

    int i_sub_1 = clip(i - 1, 0, N - 1);
    int i_add_1 = clip(i + 1, 0, N - 1);
    int j_sub_1 = clip(j - 1, 0, N - 1);
    int j_add_1 = clip(j + 1, 0, N - 1);

    if(threadIdx.x == 0){
        NW_value = side_frontiers.L_frontier[i_sub_1];
        W_value  = side_frontiers.L_frontier[i];
        SW_value = side_frontiers.L_frontier[i_add_1];

        NE_value = atomicAdd(&N_frontier[j_add_1], 0);
        E_value  = atomicAdd(&matrix[i][j_add_1], 0);

        if(i == end - 1){
            S_value  = S_frontier[j];
            SE_value = S_frontier[j_add_1];
        }
        else{
            S_value  = atomicAdd(&matrix[i_add_1][j], 0);
            SE_value = atomicAdd(&matrix[i_add_1][j_add_1], 0);
        }
    }
    else if(threadIdx.x == blockDim.x - 1){
        NE_value = side_frontiers.R_frontier[i_sub_1];
        E_value  = side_frontiers.R_frontier[i];
        SE_value = side_frontiers.R_frontier[i_add_1];

        NW_value = atomicAdd(&N_frontier[j_sub_1], 0);
        W_value  = atomicAdd(&matrix[i][j_sub_1], 0);

        if(i == end - 1){
            S_value  = S_frontier[j];
            SW_value = S_frontier[j_sub_1]; 
        }
        else{
            S_value  = atomicAdd(&matrix[i_add_1][j], 0);
            SW_value = atomicAdd(&matrix[i_add_1][j_sub_1], 0);
        }
    }
    else{
        NW_value = atomicAdd(&N_frontier[j_sub_1], 0);
        NE_value = atomicAdd(&N_frontier[j_add_1], 0);

        W_value = atomicAdd(&matrix[i][j_sub_1], 0);
        E_value = atomicAdd(&matrix[i][j_add_1], 0);

        if(i == end - 1){
            SW_value = S_frontier[j_sub_1];
            S_value  = S_frontier[j];
            SE_value = S_frontier[j_add_1];
        }
        else{
            SW_value = atomicAdd(&matrix[i_add_1][j_sub_1], 0);
            S_value  = atomicAdd(&matrix[i_add_1][j], 0);
            SE_value = atomicAdd(&matrix[i_add_1][j_add_1], 0);
        }
    }

    __syncthreads();
    
    // convolution operation
    atomicExch(&matrix[i][j], (
        NW_value * kernel[0][0] +
        N_value  * kernel[0][1] +
        NE_value * kernel[0][2] +

        W_value      * kernel[1][0] +
        center_value * kernel[1][1] +
        E_value      * kernel[1][2] + 

        SW_value * kernel[2][0] +
        S_value  * kernel[2][1] +
        SE_value * kernel[2][2]
        ) / 9
    );

    // make sure all the threads have updated the auxiliary arrays before starting next block
    atomicExch(&N_frontier[j], center_value);

    __syncthreads();
    
    // block is done with computation, increment current_line and the next block should start
    if(threadIdx.x == 0){
        atomicAdd(current_line, 1);
    }
}

int main(int argc, char ** argv){
/*
arg 1 input matrix
arg 2 kernel
arg 3 number of threads
arg 4 width and height of input matrix
arg 5 correct output file
arg 6 output file
*/
    if(argc < 7){
		std::cerr << "not enough args" << std::endl;
		exit(1);
	}
	std::ifstream f(argv[1]);
	if(!f.is_open()){
		std::cerr << "could not open file " << argv[1] << std::endl;
		exit(1);
	}
	std::ifstream k(argv[2]);
	if(!k.is_open()){
		std::cerr << "could not open file " << argv[2] << std::endl;
		exit(1);
	}

    const int P = atoi(argv[3]);
    const int N = atoi(argv[4]);

    int ** F = new int * [N];
    int ** K = new int * [KERNEL_SIZE];

    // read input and kernel matrix
    for(int i = 0; i < N; i++){
        F[i] = new int [N];
        for(int j = 0; j < N; j++){
            f >> F[i][j];
        }
    }
    for(int i = 0; i < KERNEL_SIZE; i++){
        K[i] = new int [KERNEL_SIZE];
        for(int j = 0; j < KERNEL_SIZE; j++){
            k >> K[i][j];
        }
    }
    f.close();
    k.close();

    // allocate memory and copy data to GPU
    int ** device_F;
    int ** device_K;
    int ** host_pointers_F; /* arrays on the host to */
    int ** host_pointers_K; /* store device pointers */
    host_pointers_F = new int * [N];
    host_pointers_K = new int * [KERNEL_SIZE];
    cudaCheckError(hipMalloc((void ***)&device_F, N * sizeof(int *)));
    cudaCheckError(hipMalloc((void ***)&device_K, KERNEL_SIZE * sizeof(int *)));
    for(int i = 0; i < N; i++){
        cudaCheckError(hipMalloc((void **)&host_pointers_F[i], N * sizeof(int *)));
        cudaCheckError(hipMemcpy(host_pointers_F[i], F[i], N * sizeof(int), hipMemcpyHostToDevice));
    }
    cudaCheckError(hipMemcpy(device_F, host_pointers_F, N * sizeof(int *), hipMemcpyHostToDevice));
    for(int i = 0; i < KERNEL_SIZE; i++){
        cudaCheckError(hipMalloc((void **)&host_pointers_K[i], KERNEL_SIZE * sizeof(int *)));
        cudaCheckError(hipMemcpy(host_pointers_K[i], K[i], KERNEL_SIZE * sizeof(int), hipMemcpyHostToDevice));
    }
    cudaCheckError(hipMemcpy(device_K, host_pointers_K, KERNEL_SIZE * sizeof(int *), hipMemcpyHostToDevice));
    
    int columns_remaining = N;
    int column = 0;
    std::vector<side_frontiers> side_frontiers_vector;
    while(columns_remaining > 0){
        side_frontiers side_frontier;
        int * device_Lfrontier;
        int * device_Rfrontier;
        int * host_Lfrontier = new int [N];
        int * host_Rfrontier = new int [N];

        int Lcolumn_index = clip(column - 1, 0, N - 1);
        int Rcolumn_index = clip(column + MAX_THREADS_PER_BLOCK, 0, N -1);

        // std::cout << Lcolumn_index << " " << Rcolumn_index << std::endl;
        
        for(int i = 0; i < N; i++){
            host_Lfrontier[i] = F[i][Lcolumn_index];
            host_Rfrontier[i] = F[i][Rcolumn_index];
        }

        cudaCheckError(hipMalloc((void **)&device_Lfrontier, N * sizeof(int)));
        cudaCheckError(hipMalloc((void **)&device_Rfrontier, N * sizeof(int)));

        cudaCheckError(hipMemcpy(device_Lfrontier, host_Lfrontier, N * sizeof(int), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(device_Rfrontier, host_Rfrontier, N * sizeof(int), hipMemcpyHostToDevice));

        side_frontier.L_frontier = device_Lfrontier;
        side_frontier.R_frontier = device_Rfrontier;
        side_frontiers_vector.push_back(side_frontier);

        delete host_Lfrontier;
        delete host_Rfrontier;

        column += MAX_THREADS_PER_BLOCK;
        columns_remaining -= MAX_THREADS_PER_BLOCK;
    }

    // arrays to store device pointers to auxiliary arrays and sync mechanisms
    int ** N_frontiers   = new int * [P];
    int ** S_frontiers   = new int * [P];
    int ** current_lines = new int * [P * side_frontiers_vector.size()];

    int rest = N % P;
    auto t_start = std::chrono::high_resolution_clock::now();
    
    // convolution computation
    for(int i = 0; i < P; i++){

        int start = i * (N / P) + (N % P - rest);
        int end = start + N / P;
        if(rest > 0){
            end++;
            rest--;
        }

        int * N_frontier;
        int * S_frontier;
        int * current_line;
        int host_init_current_line_value = start;

        // allocate and initialize auxiliary arrays
        cudaCheckError(hipMalloc((void **)&N_frontier, N * sizeof(int)));
        cudaCheckError(hipMalloc((void **)&S_frontier, N * sizeof(int)));
        cudaCheckError(hipMemcpy(N_frontier, F[clip(start - 1, 0, N - 1)], N * sizeof(int), hipMemcpyHostToDevice));
        cudaCheckError(hipMemcpy(S_frontier, F[clip(end, 0, N - 1)], N * sizeof(int), hipMemcpyHostToDevice));

        // store allocated pointers
        N_frontiers[i] = N_frontier;
        S_frontiers[i] = S_frontier;

        int columns_left = N;
        int side_frontier_index = 0;
        while(columns_left > 0){

            // allocate and initialize inter-block synchronization tools
            cudaCheckError(hipMalloc((void **)&current_line, sizeof(int)));
            cudaCheckError(hipMemcpy(current_line, &host_init_current_line_value, sizeof(int), hipMemcpyHostToDevice));
            // save in array for later de-allocation
            current_lines[i * side_frontiers_vector.size() + side_frontier_index] = current_line;

            int grid_index = side_frontier_index;
            if(columns_left >= MAX_THREADS_PER_BLOCK){
                convolution<<<end-start, MAX_THREADS_PER_BLOCK>>>(
                    N, start, end,
                    device_F, device_K,
                    N_frontier, S_frontier,
                    current_line,
                    side_frontiers_vector[side_frontier_index],
                    grid_index
                );
            }
            else{
                convolution<<<end-start, columns_left>>>(
                    N, start, end,
                    device_F, device_K,
                    N_frontier, S_frontier,
                    current_line,
                    side_frontiers_vector[side_frontier_index],
                    grid_index
                );
            }
        
            hipError_t hipError_t = hipGetLastError();
            if(hipError_t != hipSuccess){
                std::cerr << hipGetErrorName(hipError_t) << " " << hipGetErrorString(hipError_t);
            }

            columns_left -= MAX_THREADS_PER_BLOCK;
            side_frontier_index++;
        }
    }
    hipDeviceSynchronize();
    for(int i = 0; i < N; i++){
        cudaCheckError(hipMemcpy(F[i], host_pointers_F[i], N * sizeof(int), hipMemcpyDeviceToHost));
    }

    // get elapsed time
    auto t_end = std::chrono::high_resolution_clock::now();
    double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    std::cout << elapsed_time_ms << std::endl;

    // write to output file
    std::ofstream g(argv[6]);
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            g << F[i][j] << " ";
        }
        g << std::endl;
    }
    g.close();

    // free memory
    // matrix
    for(int i = 0; i < N; i++){
        cudaCheckError(hipFree(host_pointers_F[i]));
        delete F[i];
    }
    cudaCheckError(hipFree(device_F));
    delete host_pointers_F;
    delete F;
    // kernel
    for(int i = 0; i < KERNEL_SIZE; i++){
        cudaCheckError(hipFree(host_pointers_K[i]));
        delete K[i];
    }
    cudaCheckError(hipFree(device_K));
    delete host_pointers_K;
    delete K;
    // auxiliary arrays and sync mechanisms
    for(int i = 0; i < P; i++){
        cudaCheckError(hipFree(N_frontiers[i]));
        cudaCheckError(hipFree(S_frontiers[i]));
    }
    for(int i = 0; i < P * side_frontiers_vector.size(); i++){
        cudaCheckError(hipFree(current_lines[i]));
    }
    delete N_frontiers;
    delete S_frontiers;
    delete current_lines;
    for(int i = 0; i < side_frontiers_vector.size(); i++){
        cudaCheckError(hipFree(side_frontiers_vector[i].L_frontier));
        cudaCheckError(hipFree(side_frontiers_vector[i].R_frontier));
    }

    // error check
    std::ifstream correct(argv[5]);
    if(!correct.is_open()){
		std::cerr << "could not open file " << argv[5] << std::endl;
		exit(1);
	}
    std::ifstream output(argv[6]);
    if(!output.is_open()){
		std::cerr << "could not open file " << argv[6] << std::endl;
		exit(1);
	}
    int x,y;
    while(correct >> x){
        if(output >> y){
            if(x != y){
                std::cerr << "Incorrect output!" << std::endl;
                std::cerr << "Diff found: " << x << " " << y << std::endl;
                exit(1);
            }
        }
        else{
            std::cerr << "Incorrect output!" << std::endl;
            std::cerr << "Not enough elements in output" << std::endl;
            exit(1);
        }
    }
    if(output >> y){
        std::cerr << "Incorrect output!" << std::endl;
        std::cerr << "Too many elements in output" << std::endl;
        exit(1);
    }
    std::cout << "Correct output!" << std::endl;

    return 0;
}
